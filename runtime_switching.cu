#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "hip/hip_runtime.h"
#include ""
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvToolsExt.h>
#include <thrust/host_vector.h>
#include <tuple>
#include "types.h"
#include "runtime_switching.h"

__global__ void markTargetNodes(Node* nodes, int N, int target, int* node_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = 0;
	Node node = nodes[idx];
	if (node.depth > target)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		Node parentnode = nodes[node.parent];
		if (parentnode.depth > target)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}
	node_counts[idx] = count;
}

__global__ void putRenderIndices(Node* nodes, int N, int* node_counts, int* node_offsets, int* render_indices, int* parent_indices = nullptr, int* nodes_for_render_indices = nullptr)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];
	int count = node_counts[idx];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];
	int start = node.start;
	
	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		if (parent_indices)
			parent_indices[offset + i] = parentgaussian; 
		if (nodes_for_render_indices)
			nodes_for_render_indices[offset + i] = idx;
	}
}

int Switching::expandToTarget(
	int N,
	int target,
	int* nodes,
	int* render_indices
)
{
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);

	int num_blocks = (N + 255) / 256;
	markTargetNodes << <num_blocks, 256 >> > ((Node*)nodes, N, target, render_counts.data().get());

	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);

	putRenderIndices << <num_blocks, 256 >> > ((Node*)nodes, N, render_counts.data().get(), render_offsets.data().get(), render_indices);

	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

__device__ bool inboxCUDA(Box& box, Point viewpoint)
{
	bool inside = true;
	for (int i = 0; i < 3; i++)
	{
		inside &= viewpoint.xyz[i] >= box.minn.xyz[i] && viewpoint.xyz[i] <= box.maxx.xyz[i];
	}
	return inside;
}

__device__ float pointboxdistCUDA(Box& box, Point viewpoint)
{
	Point closest = {
		max(box.minn.xyz[0], min(box.maxx.xyz[0], viewpoint.xyz[0])),
		max(box.minn.xyz[1], min(box.maxx.xyz[1], viewpoint.xyz[1])),
		max(box.minn.xyz[2], min(box.maxx.xyz[2], viewpoint.xyz[2]))
	};

	Point diff = {
		viewpoint.xyz[0] - closest.xyz[0],
		viewpoint.xyz[1] - closest.xyz[1],
		viewpoint.xyz[2] - closest.xyz[2]
	};

	return sqrt(diff.xyz[0] * diff.xyz[0] + diff.xyz[1] * diff.xyz[1] + diff.xyz[2] * diff.xyz[2]);
}

__device__ float computeSizeGPU(Box& box, Point viewpoint, Point zdir)
{
	if (inboxCUDA(box, viewpoint))
		return FLT_MAX;

	float min_dist = pointboxdistCUDA(box, viewpoint);

	return box.maxx.xyz[3] / min_dist;
}

__global__ void changeNodesOnce(
	Node* nodes,
	int N,
	int* indices,
	Box* boxes,
	Point* viewpoint,
	Point zdir,
	float target_size,
	int* split,
	int* node_counts,
	int* node_ids,
	char* needs_children
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 1; // repeat yourself
	char need_child = 0;
	if (size >= target_size)
	{
		if (node.depth > 0 && split[node_id] == 0) // split
		{
			if (node.start_children == -1)
			{
				node_ids[idx] = node_id;
				need_child = 1;
			}
			else
			{
				count += node.count_children;
				split[node_id] = 1;
			}
		}
	}
	else
	{
		int parent_node_id = node.parent;
		if (parent_node_id != -1)
		{
			Node parent_node = nodes[parent_node_id];
			float parent_size = computeSizeGPU(boxes[parent_node_id], *viewpoint, zdir);
			if (parent_size < target_size) // collapse
			{
				split[parent_node_id] = 0;
				count = 0; // forget yourself
			}
		}
	}
	needs_children[idx] = need_child;
	node_counts[idx] = count;
}

__global__ void putNodes(
	Node* nodes,
	int N,
	int* indices,
	int* node_counts,
	int* node_offsets,
	int* next_nodes)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = node_counts[idx];
	if (count == 0)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];

	next_nodes[offset] = node_id;
	for (int i = 1; i < count; i++)
		next_nodes[offset + i] = node.start_children + i - 1;
}

__global__ void countRenderIndicesIndexed(Node* nodes, int* split, int N, int* node_indices, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = node.count_leafs;
	if (node.depth > 0 && split[node_idx] == 0)
		count += node.count_merged;

	render_counts[idx] = count;
}

__global__ void putRenderIndicesIndexed(Node* nodes, int N, int* node_indices, int* render_counts, int* render_offsets, int* render_indices, int* parent_indices, int* nodes_of_render_indices, Box* boxes, float3* debug)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = render_counts[idx];
	int offset = idx == 0 ? 0 : render_offsets[idx - 1];
	int start = node.start;

	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		parent_indices[offset + i] = parentgaussian;
		nodes_of_render_indices[offset + i] = node_idx;
	}

	if (debug != nullptr)
	{
		Box box = boxes[node_idx];
		for (int i = 0; i < count; i++)
		{
			float red = min(1.0f, node.depth / 10.0f);
			debug[offset + i] = { red, 1.0f - red, 0 };
			if (node.depth == 0)
				debug[offset + i] = { 0, 0, 1.0f };
		}
	}
}

void Switching::changeToSizeStep(
	float target_size,
	int N,
	int* node_indices,
	int* new_node_indices,
	int* nodes,
	float* boxes,
	float* viewpoint,
	float x, float y, float z,
	int* split,
	int* render_indices,
	int* parent_indices,
	int* nodes_of_render_indices,
	int* nodes_to_expand,
	float* debug,
	char*& scratchspace,
	size_t& scratchspacesize,
	int* NsrcI,
	int* NdstI,
	char* NdstC,
	int* numI,
	int maxN,
	int& add_success,
	int* new_N,
	int* new_R,
	int* need_expansion,
	void* maintenanceStream)
{
	hipStream_t stream = (hipStream_t)maintenanceStream;

	int num_node_blocks = (N + 255) / 256;

	Point zdir = { x, y, z };

	int* num_to_expand = numI;
	int* node_counts = NsrcI, * node_offsets = NdstI, * node_ids = NdstI;
	char* need_children = NdstC;
	if (scratchspacesize == 0)
	{
		size_t testsize;

		hipcub::DeviceScan::InclusiveSum(nullptr, testsize, node_counts, node_offsets, maxN, stream);
		scratchspacesize = testsize;
		hipcub::DeviceSelect::Flagged(nullptr, testsize, node_ids, need_children, nodes_to_expand, num_to_expand, maxN, stream);
		scratchspacesize = std::max(testsize, scratchspacesize);

		if (scratchspace)
			hipFree(scratchspace);
		scratchspacesize = testsize;
		hipMalloc(&scratchspace, scratchspacesize);
	}

	changeNodesOnce << <num_node_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		N, 
		node_indices, 
		(Box*)boxes, 
		(Point*)viewpoint, 
		zdir, 
		target_size, 
		split, 
		node_counts, 
		node_ids, 
		need_children
		);

	hipcub::DeviceSelect::Flagged(scratchspace, scratchspacesize, node_ids, need_children, nodes_to_expand, num_to_expand, N, stream);
	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, node_counts, node_offsets, N, stream);

	hipMemcpyAsync(need_expansion, num_to_expand, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(new_N, node_offsets + N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	if (*new_N > maxN)
	{
		add_success = 0;
		return;
	}

	putNodes << <num_node_blocks, 256, 0, stream>> > (
		(Node*)nodes,
		N, 
		node_indices, 
		node_counts, 
		node_offsets, 
		new_node_indices
		);

	int num_render_blocks = (*new_N + 255) / 256;
	int* render_counts = NsrcI, * render_offsets = NdstI;

	countRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		split, 
		*new_N, 
		new_node_indices, 
		render_counts
		);

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, render_counts, render_offsets, *new_N, stream);

	putRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		*new_N, 
		new_node_indices, 
		render_counts, 
		render_offsets, 
		render_indices, 
		parent_indices, 
		nodes_of_render_indices, 
		(Box*)boxes,
		(float3*)debug
		);

	hipMemcpyAsync(new_R, render_offsets + *new_N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);

	add_success = 1;
}

__global__ void markNodesForSize(Node* nodes, Box* boxes, int N, Point* viewpoint, Point zdir, float target_size, int* render_counts, int* node_markers)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = idx;
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 0;
	if (size >= target_size)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		float parent_size = computeSizeGPU(boxes[node.parent], *viewpoint, zdir);
		if (parent_size >= target_size)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}

	if (count != 0 && node_markers != nullptr)
		node_markers[node_id] = 1;

	if (render_counts != nullptr)
		render_counts[node_id] = count;
}

__global__ void computeTsIndexed(
	Node* nodes,
	Box* boxes,
	int N,
	int* indices,
	Point viewpoint,
	Point zdir,
	float target_size,
	float* ts,
	int* kids
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];

	float t;
	if (node.parent == -1)
		t = 1.0f;
	else
	{
		float parentsize = computeSizeGPU(boxes[node.parent], viewpoint, zdir);

		if (parentsize > 2.0f * target_size)
			t = 1.0f;
		else
		{
			float size = computeSizeGPU(boxes[node_id], viewpoint, zdir);
			float start = max(0.5f * parentsize, size);
			float diff = parentsize - start;

			if (diff <= 0)
				t = 1.0f;
			else
			{
				float tdiff = max(0.0f, target_size - start);
				t = max(1.0f - (tdiff / diff), 0.0f);
			}
		}
	}

	ts[idx] = t;
	kids[idx] = (node.parent == -1) ? 1 : nodes[node.parent].count_children;
}

void Switching::getTsIndexed(
	int N,
	int* indices,
	float target_size,
	int* nodes,
	float* boxes,
	float vx, float vy, float vz,
	float x, float y, float z,
	float* ts,
	int* kids,
	void* stream
)
{
	Point zdir = { x, y, z };
	Point cam = { vx, vy, vz };
	int num_blocks = (N + 255) / 256;
	computeTsIndexed<<<num_blocks, 256, 0, (hipStream_t)stream >>>(
		(Node*)nodes, 
		(Box*)boxes, 
		N, 
		indices, 
		cam,
		zdir, 
		target_size, 
		ts, 
		kids);
}

int Switching::expandToSize(
	int N,
	float target_size,
	int* nodes,
	float* boxes,
	float* viewpoint,
	float x, float y, float z,
	int* render_indices,
	int* node_markers,
	int* parent_indices,
	int* nodes_for_render_indices)
{
	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);

	Point zdir = { x, y, z };

	int num_blocks = (N + 255) / 256;
	markNodesForSize << <num_blocks, 256 >> > ((Node*)nodes, (Box*)boxes, N, (Point*)viewpoint, zdir, target_size, render_counts.data().get(), node_markers);

	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);

	putRenderIndices << <num_blocks, 256 >> > ((Node*)nodes, N, render_counts.data().get(), render_offsets.data().get(), render_indices, parent_indices, nodes_for_render_indices);

	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

void Switching::markVisibleForAllViewpoints(
	float target_size,
	int* nodes,
	int num_nodes,
	float* boxes,
	float* viewpoints,
	int num_viewpoints,
	int* seen,
	float zx,
	float zy,
	float zz
)
{
	thrust::device_vector<int> seen_cuda(num_nodes);
	thrust::device_vector<Point> viewpoint_cuda(1);
	thrust::device_vector<Node> nodes_cuda(num_nodes);
	thrust::device_vector<Box> boxes_cuda(num_nodes);

	hipMemcpy(nodes_cuda.data().get(), nodes, sizeof(Node) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(boxes_cuda.data().get(), boxes, sizeof(Box) * num_nodes, hipMemcpyHostToDevice);

	Point zdir = { zx, zy, zz };

	Point* points = (Point*)viewpoints;
	int num_blocks = (num_nodes + 255) / 256;
	for (int i = 0; i < num_viewpoints; i++)
	{
		Point viewpoint = points[i];
		hipMemcpy(viewpoint_cuda.data().get(), &viewpoint, sizeof(Point), hipMemcpyHostToDevice);

		markNodesForSize << <num_blocks, 256 >> > (
			nodes_cuda.data().get(),
			boxes_cuda.data().get(),
			num_nodes,
			viewpoint_cuda.data().get(),
			zdir,
			target_size,
			nullptr,
			seen_cuda.data().get());
	}
	hipMemcpy(seen, seen_cuda.data().get(), sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

	if (hipDeviceSynchronize())
		std::cout << "Errors: " << hipDeviceSynchronize() << std::endl;
}